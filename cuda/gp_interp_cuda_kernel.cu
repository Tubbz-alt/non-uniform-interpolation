#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
// #include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_gaussian_density(
    scalar_t x, scalar_t y,
    scalar_t mean_x, scalar_t mean_y,
    scalar_t std_x, scalar_t std_y) {

    const auto x_exp_term = (x - mean_x) / std_x;
    const auto y_exp_term = (y - mean_y) / std_y;

    const auto exp_term = __expf(-0.5 * (x_exp_term * x_exp_term + y_exp_term * y_exp_term));
    const auto std_term = 1.0 / (std_x * std_y);
    const auto const_term = 1.0 / (2.0 * 3.141592654);

    return exp_term * std_term * const_term;
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t d_normal_pdf_d_mu_i(
    scalar_t x, scalar_t mean_x, scalar_t std_x, scalar_t density_value) {

    return density_value * (x - mean_x) / (std_x * std_x);
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t d_normal_pdf_d_std_i(
    scalar_t x, scalar_t mean_x, scalar_t std_x, scalar_t density_value) {
    return -density_value * (x - mean_x) * (x - mean_x) / (std_x * std_x * std_x * std_x);
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_distance_weight(
    scalar_t x, scalar_t y,
    scalar_t mean_x, scalar_t mean_y) {

    const auto x_diff = (x - mean_x);
    const auto y_diff = (y - mean_y);
    const auto dist2 = x_diff * x_diff + y_diff * y_diff;
    const auto weight = 1.0 / dist2;

    return weight;
}


template <typename scalar_t>
__device__ scalar_t clamp(scalar_t x, scalar_t min_val, scalar_t max_val) {
    return max(min_val, min(max_val, x));
}



template <typename scalar_t>
__global__ void gp_interp_compute_color_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> image,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> means,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> stds,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights) {

    const int point_idx = blockDim.y * blockIdx.y + blockIdx.x;
    const int radius = blockDim.x / 2;
    const int center_x = static_cast<int>(clamp(round(means[point_idx][0]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(2))));
    const int center_y = static_cast<int>(clamp(round(means[point_idx][1]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(1))));
    const int shift_x = threadIdx.x - radius;
    const int shift_y = threadIdx.y - radius;
    const int pixel_pos_x = center_x + shift_x;
    const int pixel_pos_y = center_y + shift_y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < image.size(2) && pixel_pos_y < image.size(1)) {
        scalar_t weight = compute_gaussian_density(
            static_cast<scalar_t>(pixel_pos_x),
            static_cast<scalar_t>(pixel_pos_y),
            means[point_idx][0],
            means[point_idx][1],
            stds[point_idx][0],
            stds[point_idx][1]);
        // scalar_t weight = compute_distance_weight(
        //     static_cast<scalar_t>(pixel_pos_x),
        //     static_cast<scalar_t>(pixel_pos_y),
        //     means[point_idx][0],
        //     means[point_idx][1]);

        if (weight > 0.0) {
            for (int c = 0; c < image.size(0); c++) {
                const scalar_t color_value = image[c][center_y][center_x]; // TODO: keep the color in the shared memory
                atomicAdd(&output_image[c][pixel_pos_y][pixel_pos_x], color_value * weight);
            }

            atomicAdd(&pixel_weights[pixel_pos_y][pixel_pos_x], weight);
        }
    }
}


template <typename scalar_t>
__global__ void gp_interp_normalize_color_kernel(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights) {

    const int pixel_pos_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int pixel_pos_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < output_image.size(2) && pixel_pos_y < output_image.size(1)) {
        const scalar_t weight = pixel_weights[pixel_pos_y][pixel_pos_x];

        if (weight > 0.0) {
            for (int c = 0; c < output_image.size(0); c++) {
                output_image[c][pixel_pos_y][pixel_pos_x] /= weight;
            }
        }
    }
}


template <typename scalar_t>
__global__ void gp_interp_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_output_image,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> means,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> stds,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_means,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_stds) {

    const int point_idx = blockDim.y * blockIdx.y + blockIdx.x;
    const int radius = blockDim.x / 2;
    const int center_x = static_cast<int>(clamp(round(means[point_idx][0]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(2))));
    const int center_y = static_cast<int>(clamp(round(means[point_idx][1]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(1))));
    const int shift_x = threadIdx.x - radius;
    const int shift_y = threadIdx.y - radius;
    const int pixel_pos_x = center_x + shift_x;
    const int pixel_pos_y = center_y + shift_y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < image.size(2) && pixel_pos_y < image.size(1)) {
        scalar_t weight = compute_gaussian_density(
            static_cast<scalar_t>(pixel_pos_x),
            static_cast<scalar_t>(pixel_pos_y),
            means[point_idx][0],
            means[point_idx][1],
            stds[point_idx][0],
            stds[point_idx][1]);
        scalar_t total_weight = pixel_weights[pixel_pos_y][pixel_pos_x];

        if (weight > 0.0 && total_weight > 0.0) {
            scalar_t d_v_d_mu_x = 0.0;
            scalar_t d_v_d_mu_y = 0.0;
            scalar_t d_v_d_std_x = 0.0;
            scalar_t d_v_d_std_y = 0.0;

            scalar_t d_weight_d_mu_x = (1.0 / total_weight - weight) * d_normal_pdf_d_mu_i(
                static_cast<scalar_t>(pixel_pos_x), means[point_idx][0], stds[point_idx][0], weight);
            scalar_t d_weight_d_mu_y = (1.0 / total_weight - weight) * d_normal_pdf_d_mu_i(
                static_cast<scalar_t>(pixel_pos_y), means[point_idx][1], stds[point_idx][1], weight);
            scalar_t d_weight_d_std_x = (1.0 / total_weight - weight) * d_normal_pdf_d_std_i(
                static_cast<scalar_t>(pixel_pos_x), means[point_idx][0], stds[point_idx][0], weight);
            scalar_t d_weight_d_std_y = (1.0 / total_weight - weight) * d_normal_pdf_d_std_i(
                static_cast<scalar_t>(pixel_pos_y), means[point_idx][1], stds[point_idx][1], weight);

            for (int c = 0; c < image.size(0); c++) {
                const scalar_t color_value = image[c][center_y][center_x]; // TODO: keep the color in the shared memory
                d_v_d_mu_x += color_value * d_weight_d_mu_x;
                d_v_d_mu_y += color_value * d_weight_d_mu_y;
                d_v_d_std_x += color_value * d_weight_d_std_x;
                d_v_d_std_y += color_value * d_weight_d_std_y;
            }

            atomicAdd(&grad_means[point_idx][0], d_v_d_mu_x);
            atomicAdd(&grad_means[point_idx][1], d_v_d_mu_y);
            atomicAdd(&grad_stds[point_idx][0], d_v_d_std_x);
            atomicAdd(&grad_stds[point_idx][1], d_v_d_std_y);
        }
    }
}
} // namespace

std::vector<torch::Tensor> gp_interp_cuda_forward(
    torch::Tensor image,
    torch::Tensor means,
    torch::Tensor stds,
    int radius) {

    const auto num_points = means.size(0);
    auto output_image = torch::zeros_like(image).contiguous();
    auto pixel_weights = torch::zeros({image.size(1), image.size(2)}).to(output_image.device()).contiguous();

    {
        const dim3 threads(radius * 2, radius * 2);
        const int blocks = num_points;

        AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_compute_color_kernel", ([&] {
        gp_interp_compute_color_kernel<scalar_t><<<blocks, threads>>>(
            image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
        }));
        AT_CUDA_CHECK(hipGetLastError());

        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        // }
    }

    {
        const dim3 threads(16, 16);
        const dim3 blocks((image.size(2) + 16 - 1) / 16, (image.size(1) + 16 - 1) / 16);
        AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_normalize_color_kernel", ([&] {
        gp_interp_normalize_color_kernel<scalar_t><<<blocks, threads>>>(
            output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
        }));
        AT_CUDA_CHECK(hipGetLastError());

        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        // }
    }

    return {output_image, pixel_weights};
}


std::vector<torch::Tensor> gp_interp_cuda_backward(
    torch::Tensor grad_output_image,
    torch::Tensor image,
    torch::Tensor means,
    torch::Tensor stds,
    int radius,
    torch::Tensor output_image,
    torch::Tensor pixel_weights) {

    auto grad_image = torch::zeros_like(image).contiguous();
    auto grad_means = torch::zeros_like(means).contiguous();
    auto grad_stds = torch::zeros_like(stds).contiguous();

    const dim3 threads(radius * 2, radius * 2);
    const auto num_points = means.size(0);
    const int blocks = num_points;

    AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_cuda_backward", ([&] {
    gp_interp_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        grad_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        grad_means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        grad_stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    }));
    AT_CUDA_CHECK(hipGetLastError());

    return {grad_image, grad_means, grad_stds};
}
